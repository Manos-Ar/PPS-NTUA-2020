#include "hip/hip_runtime.h"
/*
 *  dmm_gpu.cu -- Template for DMM GPU kernels
 *
 *  Copyright (C) 2020, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2020, Athena Elafrou/Petros Anastasiadis
 */

#include <stdio.h>

#include "hipblas.h"
#include "dmm.h"

/*
 *  Naive kernel
 */
__global__ void dmm_gpu_naive(const value_t *A, const value_t *B, value_t *C,
                              const size_t M, const size_t N, const size_t K) {
  // Compute the row and the column of the current thread
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  value_t Cvalue = 0;

  // If the threads positions is out of array bounds, exit
  if (row >= M || col >= N) return;

  // Each thread computes one element of C by accumulating results into Cvalue
  for (int e = 0; e < K; e++) {
    Cvalue += A[row * K + e] * B[e * N + col];
  }

  C[row * N + col] = Cvalue;
}

/*
 *  Coalesced memory acceses of A.
 */
__global__ void dmm_gpu_coalesced_A(const value_t *A, const value_t *B,
                                    value_t *C, const size_t M, const size_t N,
                                    const size_t K) {
  //   // Compute the Block row and column of each thread
  //     int blockRow = blockIdx.y;
  //   int blockCol = blockIdx.x;

  //   // Each thread block computes one sub-matrix Csub of C

  //   // Each thread computes one element of C by accumulating results into
  //   Cvalue value_t Cvalue;

  //   // Thread row and column within Csub
  //   int row = threadIdx.y;
  //   int col = threadIdx.x;

  //   // If the threads positions is out of array bounds, exit
  //   if (row >= M || col >= N) return;

  //   // Loop over all the sub-matrices of A and B that are required to compute
  //   Csub
  //   // Multiply each pair of sub-matrices together and accumulate the results
  //   for (int m = 0; m < (K / BLOCK_SIZE); ++m) {
  //     // Get sub-matrix Asub of A

  //     // Get sub-matrix Bsub of B

  //     // Shared memory used to store Asub and Bsub respectively
  //     __shared__ value_t As[BLOCK_SIZE][BLOCK_SIZE];
  //     __shared__ value_t As[BLOCK_SIZE][BLOCK_SIZE];
  //   }

  //   // Each thread computes one element of C by accumulating results into
  //   Cvalue for (int e = 0; e < K; e++) {
  //     Cvalue += A[row * K + e] * B[e * N + col];
  //   }

  //   C[row * N + col] = Cvalue;
}

/*
 *  Reduced memory accesses.
 */
__global__ void dmm_gpu_reduced_global(const value_t *A, const value_t *B,
                                       value_t *C, const size_t M,
                                       const size_t N, const size_t K) {
  /*
   * FILLME: fill the code.
   */
}

/*
 *  Use of cuBLAS
 */
void dmm_gpu_cublas(const value_t *A, const value_t *B, value_t *C,
                    const size_t M, const size_t N, const size_t K) {
  // Define variables for cuBLAS status and handle
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  // Define leading dimensions of A,B,C matrices for hipblasSgemm
  int lda = N;
  int ldb = K;
  int ldc = N;

  // Define alpha, beta values for GEMM calculation
  // C = alpha*A*B + beta*C
  const value_t alpha_val = 1;
  const value_t beta_val = 0;
  const value_t *alpha = &alpha_val;
  const value_t *beta = &beta_val;

  // Create a handle for cuBLAS
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS initialization failed\n");
  }

  // Call hipblasSgemm to calculate the DMM (for floats)
  stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, alpha, A, lda,
                     B, ldb, beta, C, ldc);

  // Call hipblasSgemm to calculate the DMM (for doubles)
  // stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, alpha, A,
  //   lda, B, ldb, beta, C, ldc);

  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("hipblasSgemm failed");
  }

  // Destroy the handle
  hipblasDestroy(handle);
}
