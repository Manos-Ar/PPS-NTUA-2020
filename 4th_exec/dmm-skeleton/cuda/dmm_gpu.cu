#include "hip/hip_runtime.h"
/*
 *  dmm_gpu.cu -- Template for DMM GPU kernels
 *
 *  Copyright (C) 2020, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2020, Athena Elafrou/Petros Anastasiadis
 */

#include <stdio.h>

#include "hipblas.h"
#include "dmm.h"

/*
 *  Naive kernel
 */
__global__ void dmm_gpu_naive(const value_t *A, const value_t *B, value_t *C,
                              const size_t M, const size_t N, const size_t K) {
  // Compute the row and the column of the current thread
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  value_t Cvalue = 0;

  // If the threads positions is out of array bounds, exit
  if (row >= M || col >= N) return;

  // Each thread computes one element of C by accumulating results into Cvalue
  for (int e = 0; e < K; e++) {
    Cvalue += A[row * K + e] * B[e * N + col];
  }

  C[row * N + col] = Cvalue;
}

/*
 *  Coalesced memory acceses of A.
 */
__global__ void dmm_gpu_coalesced_A(const value_t *A, const value_t *B,
                                    value_t *C, const size_t M, const size_t N,
                                    const size_t K) {
  // Shared memory between threads of the same block, for Tiled sub-matrix of A
  __shared__ value_t A_shared[TILE_X][TILE_Y];

  // Compute the row and column of each thread in a Tile
  int tid_y = threadIdx.y;
  int tid_x = threadIdx.x;
  int row = blockIdx.y * TILE_Y + tid_y;
  int col = blockIdx.x * TILE_X + tid_x;

  // Each thread computes one element of C by accumulating results into Cvalue
  value_t Cvalue = 0;

  // Calculate the ceiling for number of Tiles needed for A
  int tile_x_ceil = (K + TILE_X - 1) / TILE_X;

  // Loop over all the sub-matrices of A (on x-axis, for all columns) that are
  // required to compute Csub.
  for (int m = 0; m < tile_x_ceil; m++) {
    // Load sub-matrix of A from device memory to shared memory
    A_shared[tid_y][tid_x] = A[row * K + m * TILE_X + tid_x];
    // Explanation: Find from which row to start, from which tile, from which
    // thread inside the tile

    // Synchronize to make sure the sub-matrix is loaded
    // before starting the computation
    __syncthreads();

    // Multiply the A sub-matrix with B and accumulate the results.
    for (int e = 0; e < TILE_X; e++) {
      Cvalue += A_shared[tid_y][e] * B[(m * TILE_X + e) * N + col];
    }
    // Synchronize to make sure that the preceding computation is done
    // before loading new sub-matrix of A and in the next iteration
    __syncthreads();
  }
  // Write Csub to device memory
  C[row * N + col] = Cvalue;
}

/*
 *  Reduced memory accesses.
 */
__global__ void dmm_gpu_reduced_global(const value_t *A, const value_t *B,
                                       value_t *C, const size_t M,
                                       const size_t N, const size_t K) {
  // Shared memory between threads of the same block, for Tiled sub-matrices A,B
  __shared__ value_t A_shared[TILE_X][TILE_Y];
  __shared__ value_t B_shared[TILE_X][TILE_Y];

  // Compute the row and column of each thread in a Tile
  int tid_y = threadIdx.y;
  int tid_x = threadIdx.x;
  int row = blockIdx.y * TILE_Y + tid_y;
  int col = blockIdx.x * TILE_X + tid_x;

  // Each thread computes one element of C by accumulating results into Cvalue
  value_t Cvalue = 0;

  // Calculate the ceiling for number of Tiles needed for A
  int tile_x_ceil = (K + TILE_X - 1) / TILE_X;

  // Loop over all the sub-matrices of A (on x-axis, for all columns) and
  // B (on y-axis, for all rows) that are required to compute Csub.
  for (int m = 0; m < tile_x_ceil; m++) {
    // Load sub-matrices of A,B from device memory to shared memory
    A_shared[tid_y][tid_x] = A[row * K + m * TILE_X + tid_x];
    B_shared[tid_y][tid_x] = B[col + (m * TILE_Y + tid_y) * N];
    // Explanation A: Find from which row to start, from which tile, from which
    // thread inside the tile
    // Explanation B: Find from which col to start, from which tile, from which
    // thread inside the tile

    // Synchronize to make sure the sub-matrix is loaded
    // before starting the computation
    __syncthreads();

    // Multiply the sub-matrices together and accumulate the results.
    for (int e = 0; e < TILE_X; e++) {
      Cvalue += A_shared[tid_y][e] * B_shared[e][tid_x];
    }
    // Synchronize to make sure that the preceding computation is done
    // before loading new sub-matrix of A and in the next iteration
    __syncthreads();
  }
  // Write Csub to device memory
  C[row * N + col] = Cvalue;
}

/*
 *  Use of cuBLAS
 */
void dmm_gpu_cublas(const value_t *A, const value_t *B, value_t *C,
                    const size_t M, const size_t N, const size_t K) {
  // Define variables for cuBLAS status and handle
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  // Define leading dimensions of A,B,C matrices for hipblasSgemm
  int lda = N;
  int ldb = K;
  int ldc = N;

  // Define alpha, beta values for GEMM calculation
  // C = alpha*A*B + beta*C
  const value_t alpha_val = 1;
  const value_t beta_val = 0;
  const value_t *alpha = &alpha_val;
  const value_t *beta = &beta_val;

  // Create a handle for cuBLAS
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS initialization failed\n");
  }

  // Call hipblasSgemm to calculate the DMM (for floats)
  stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, alpha, A, lda,
                     B, ldb, beta, C, ldc);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("hipblasSgemm failed");
  }

  // Destroy the handle
  hipblasDestroy(handle);
}
